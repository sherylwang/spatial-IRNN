#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// SIAMESE RECURRENT ARCHITECTURE FOR VISUAL TRACKING
// Version 1.0, Copyright(c) July, 2017
// Xiaqing Xu, Bingpeng Ma, Hong Chang, Xilin Chen
// Written by Xiaqing Xu
// ------------------------------------------------------------------

#include <vector>
#include <iostream>

#include "caffe/filler.hpp"
#include "caffe/layers/spatial_irnn_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe{
template <typename Dtype>
__global__ void ReLUForward(const int n, Dtype* in) {
  CUDA_KERNEL_LOOP(index, n) {
    in[index] = in[index] > 0 ? in[index] : Dtype(0.);
  }
}

template <typename Dtype>
__global__ void ReLUBackward(const int n, Dtype* out_diff,const Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = Dtype(1.) * (top_data[index] > 0);
  }
}

template <typename Dtype>
void RNNDOWNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top){
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const int count = top[0]->count();
  const Dtype* w = this->blobs_[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data(); 

  caffe_copy(count, bottom_data, top_data);
 
  for(int i = 0; i < H_; i++){
    if(i > 0){
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, NH_, W_ * N_, NH_, Dtype(1.),
          w, top_data + (i - 1) * NH_ * N_* W_ , Dtype(1.),
          top_data + i * NH_ * N_* W_);
  }
  ReLUForward<Dtype><<<CAFFE_GET_BLOCKS(NH_*W_*N_), CAFFE_CUDA_NUM_THREADS>>>(
      NH_ * W_ * N_, top_data + i * NH_ * N_* W_);
  CUDA_POST_KERNEL_CHECK;
  }
}

template <typename Dtype>
void RNNDOWNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom){  
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const int count = bottom[0]->count();
  const Dtype* w = this->blobs_[0]->gpu_data();

  Dtype* w_diff = this->blobs_[0]->mutable_gpu_diff();
  // dh
  Dtype* h_diff = cache_.mutable_gpu_data();
  // f'(h)
  Dtype* f_diff = cache_.mutable_gpu_diff();

  Dtype* hh_diff = hh_.mutable_gpu_diff();

  ReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, f_diff, top_data);
  CUDA_POST_KERNEL_CHECK;
  
  caffe_copy(count, top_diff, h_diff);

  for(int i = H_ - 1; i >= 0; i--){
    // dzdf
    caffe_gpu_mul(NH_ * W_ * N_, h_diff + i * NH_ * N_ * W_,
    f_diff + i * NH_ * N_ * W_, f_diff + i * NH_ * N_* W_);
    // dzdhh
    caffe_gpu_gemm(CblasTrans, CblasNoTrans, NH_, W_ * N_, NH_, Dtype(1.),
        w, f_diff + i * NH_ * N_* W_ , Dtype(0.), hh_diff);
    if(i > 0){
      caffe_gpu_add(NH_ * W_ * N_, hh_diff,
          h_diff + (i - 1)* NH_ * N_ * W_,
          h_diff + (i - 1)* NH_ * N_ * W_);
      caffe_gpu_gemm(CblasNoTrans, CblasTrans, NH_, NH_, W_ * N_, Dtype(1.),
          f_diff + i * NH_ * N_ * W_ , top_data + (i - 1) * NH_ * N_ * W_,
          Dtype(1.),  w_diff);
    }
  } 

  if(propagate_down[0]){
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(bottom[0]->count(), f_diff, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(RNNDOWNLayer);
}  // namespace caffe
 


